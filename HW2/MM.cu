#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "kernel.h"

// CUDA kernel for matrix multiplication
__global__ void matrixMulKernel(float* input1, float* input2, int M, int N, int K, float* output) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute one element of the output matrix
    if (row < M && col < K) {
        float sum = 0.0f;
        for (int i = 0; i < N; ++i) {
            sum += input1[row * N + i] * input2[i * K + col];
        }
        output[row * K + col] = sum;
    }
}

void matrixMul(array2d_t<float>& input1, array2d_t<float>& input2, array2d_t<float>& output) {
    int M = input1.row_count;
    int N = input1.col_count;
    int K = input2.col_count;
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((K + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(input1.data_ptr, input2.data_ptr, M, N, K, output.data_ptr);
}